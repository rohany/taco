#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "cudalibs.h"
#include "taco_legion_header.h"
#include "taco_mapper.h"
#define TACO_MIN(_a,_b) ((_a) < (_b) ? (_a) : (_b))
using namespace Legion;
typedef FieldAccessor<READ_ONLY,double,2,coord_t,Realm::AffineAccessor<double,2,coord_t>> AccessorROdouble2;
typedef FieldAccessor<READ_WRITE,double,2,coord_t,Realm::AffineAccessor<double,2,coord_t>> AccessorRWdouble2;

struct task_1Args {
  int32_t gridX;
  int32_t gridY;
};
struct task_2Args {
  int32_t gridX;
  int32_t gridY;
};
struct task_3Args {
  int32_t gridX;
  int32_t gridY;
};
struct task_4Args {
  int32_t kios;
};
struct task_5Args {
  int64_t aPartitionBounds0hi;
  int64_t aPartitionBounds0lo;
  int64_t aPartitionBounds1hi;
  int64_t aPartitionBounds1lo;
  int32_t b1_dimension;
  int32_t b2_dimension;
  int32_t c1_dimension;
  int32_t c2_dimension;
  int32_t gridX;
  int32_t in;
  int32_t jn;
  int32_t kos;
};
struct task_6Args {
  int32_t a1_dimension;
  int32_t a2_dimension;
  int64_t aPartitionBounds0hi;
  int64_t aPartitionBounds0lo;
  int64_t aPartitionBounds1hi;
  int64_t aPartitionBounds1lo;
  int32_t b1_dimension;
  int32_t b2_dimension;
  int32_t c1_dimension;
  int32_t c2_dimension;
  int32_t gridX;
  int32_t in;
  int32_t jn;
  int32_t kos;
};
struct task_7Args {
  int32_t a1_dimension;
  int32_t a2_dimension;
  int32_t b1_dimension;
  int32_t b2_dimension;
  int32_t c1_dimension;
  int32_t c2_dimension;
  int32_t gridX;
};

void task_1(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion a = regions[0];

  int32_t distFused = task->index_point[0];
  task_1Args* args = (task_1Args*)(task->args);
  int32_t gridX = args->gridX;
  int32_t gridY = args->gridY;


  int32_t in = getIndexPoint(task, 0);
  int32_t jn = getIndexPoint(task, 1);
}

LogicalPartition placeLegionA(Context ctx, Runtime* runtime, LogicalRegion a, int32_t gridX, int32_t gridY) {
  int a1_dimension = runtime->get_index_space_domain(get_index_space(a)).hi()[0] + 1;
  int a2_dimension = runtime->get_index_space_domain(get_index_space(a)).hi()[1] + 1;
  auto a_index_space = get_index_space(a);

  Point<2> lowerBound = Point<2>(0, 0);
  Point<2> upperBound = Point<2>((gridX - 1), (gridY - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<2>(lowerBound, upperBound));
  DomainT<2> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<2>(distFusedIndexSpace));
  DomainPointColoring aColoring = DomainPointColoring();
  for (PointInDomainIterator<2> itr = PointInDomainIterator<2>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[0];
    int32_t jn = (*itr)[1];
    Point<2> aStart = Point<2>((in * ((a1_dimension + (gridX - 1)) / gridX) + 0 / gridX), (jn * ((a2_dimension + (gridY - 1)) / gridY) + 0 / gridY));
    Point<2> aEnd = Point<2>(TACO_MIN((in * ((a1_dimension + (gridX - 1)) / gridX) + ((a1_dimension + (gridX - 1)) / gridX - 1)),(a1_dimension - 1)), TACO_MIN((jn * ((a2_dimension + (gridY - 1)) / gridY) + ((a2_dimension + (gridY - 1)) / gridY - 1)),(a2_dimension - 1)));
    Rect<2> aRect = Rect<2>(aStart, aEnd);
    auto aDomain = runtime->get_index_space_domain(ctx, a_index_space);
    if (!aDomain.contains(aRect.lo) || !aDomain.contains(aRect.hi)) aRect = aRect.make_empty();

    aColoring[(*itr)] = aRect;
  }
  auto aPartition = runtime->create_index_partition(ctx, a_index_space, domain, aColoring, LEGION_COMPUTE_KIND);
  LogicalPartition aLogicalPartition = runtime->get_logical_partition(ctx, get_logical_region(a), aPartition);
  RegionRequirement aReq = RegionRequirement(aLogicalPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(a));
  aReq.add_field(FID_VAL);
  task_1Args taskArgsRaw;
  taskArgsRaw.gridX = gridX;
  taskArgsRaw.gridY = gridY;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_1Args));
  IndexLauncher launcher = IndexLauncher(taskID(1), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(aReq);
  auto fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  return runtime->get_logical_partition(ctx, get_logical_region(a), aPartition);

}

void task_2(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion b = regions[0];

  int32_t distFused = task->index_point[0];
  task_2Args* args = (task_2Args*)(task->args);
  int32_t gridX = args->gridX;
  int32_t gridY = args->gridY;


  int32_t in = getIndexPoint(task, 0);
  int32_t jn = getIndexPoint(task, 1);
}

LogicalPartition placeLegionB(Context ctx, Runtime* runtime, LogicalRegion b, int32_t gridX, int32_t gridY) {
  int b1_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[0] + 1;
  int b2_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[1] + 1;
  auto b_index_space = get_index_space(b);

  Point<2> lowerBound = Point<2>(0, 0);
  Point<2> upperBound = Point<2>((gridX - 1), (gridY - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<2>(lowerBound, upperBound));
  DomainT<2> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<2>(distFusedIndexSpace));
  DomainPointColoring bColoring = DomainPointColoring();
  for (PointInDomainIterator<2> itr = PointInDomainIterator<2>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[0];
    int32_t jn = (*itr)[1];
    Point<2> bStart = Point<2>((in * ((b1_dimension + (gridX - 1)) / gridX) + 0 / gridX), (jn * ((b2_dimension + (gridY - 1)) / gridY) + 0 / gridY));
    Point<2> bEnd = Point<2>(TACO_MIN((in * ((b1_dimension + (gridX - 1)) / gridX) + ((b1_dimension + (gridX - 1)) / gridX - 1)),(b1_dimension - 1)), TACO_MIN((jn * ((b2_dimension + (gridY - 1)) / gridY) + ((b2_dimension + (gridY - 1)) / gridY - 1)),(b2_dimension - 1)));
    Rect<2> bRect = Rect<2>(bStart, bEnd);
    auto bDomain = runtime->get_index_space_domain(ctx, b_index_space);
    if (!bDomain.contains(bRect.lo) || !bDomain.contains(bRect.hi)) bRect = bRect.make_empty();

    bColoring[(*itr)] = bRect;
  }
  auto bPartition = runtime->create_index_partition(ctx, b_index_space, domain, bColoring, LEGION_COMPUTE_KIND);
  LogicalPartition bLogicalPartition = runtime->get_logical_partition(ctx, get_logical_region(b), bPartition);
  RegionRequirement bReq = RegionRequirement(bLogicalPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(b));
  bReq.add_field(FID_VAL);
  task_2Args taskArgsRaw;
  taskArgsRaw.gridX = gridX;
  taskArgsRaw.gridY = gridY;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_2Args));
  IndexLauncher launcher = IndexLauncher(taskID(2), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(bReq);
  auto fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  return runtime->get_logical_partition(ctx, get_logical_region(b), bPartition);

}

void task_3(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion c = regions[0];

  int32_t distFused = task->index_point[0];
  task_3Args* args = (task_3Args*)(task->args);
  int32_t gridX = args->gridX;
  int32_t gridY = args->gridY;


  int32_t in = getIndexPoint(task, 0);
  int32_t jn = getIndexPoint(task, 1);
}

LogicalPartition placeLegionC(Context ctx, Runtime* runtime, LogicalRegion c, int32_t gridX, int32_t gridY) {
  int c1_dimension = runtime->get_index_space_domain(get_index_space(c)).hi()[0] + 1;
  int c2_dimension = runtime->get_index_space_domain(get_index_space(c)).hi()[1] + 1;
  auto c_index_space = get_index_space(c);

  Point<2> lowerBound = Point<2>(0, 0);
  Point<2> upperBound = Point<2>((gridX - 1), (gridY - 1));
  auto distFusedIndexSpace = runtime->create_index_space(ctx, Rect<2>(lowerBound, upperBound));
  DomainT<2> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<2>(distFusedIndexSpace));
  DomainPointColoring cColoring = DomainPointColoring();
  for (PointInDomainIterator<2> itr = PointInDomainIterator<2>(domain); itr.valid(); itr++) {
    int32_t in = (*itr)[0];
    int32_t jn = (*itr)[1];
    Point<2> cStart = Point<2>((in * ((c1_dimension + (gridX - 1)) / gridX) + 0 / gridX), (jn * ((c2_dimension + (gridY - 1)) / gridY) + 0 / gridY));
    Point<2> cEnd = Point<2>(TACO_MIN((in * ((c1_dimension + (gridX - 1)) / gridX) + ((c1_dimension + (gridX - 1)) / gridX - 1)),(c1_dimension - 1)), TACO_MIN((jn * ((c2_dimension + (gridY - 1)) / gridY) + ((c2_dimension + (gridY - 1)) / gridY - 1)),(c2_dimension - 1)));
    Rect<2> cRect = Rect<2>(cStart, cEnd);
    auto cDomain = runtime->get_index_space_domain(ctx, c_index_space);
    if (!cDomain.contains(cRect.lo) || !cDomain.contains(cRect.hi)) cRect = cRect.make_empty();

    cColoring[(*itr)] = cRect;
  }
  auto cPartition = runtime->create_index_partition(ctx, c_index_space, domain, cColoring, LEGION_COMPUTE_KIND);
  LogicalPartition cLogicalPartition = runtime->get_logical_partition(ctx, get_logical_region(c), cPartition);
  RegionRequirement cReq = RegionRequirement(cLogicalPartition, 0, READ_ONLY, EXCLUSIVE, get_logical_region(c));
  cReq.add_field(FID_VAL);
  task_3Args taskArgsRaw;
  taskArgsRaw.gridX = gridX;
  taskArgsRaw.gridY = gridY;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_3Args));
  IndexLauncher launcher = IndexLauncher(taskID(3), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(cReq);
  auto fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();
  return runtime->get_logical_partition(ctx, get_logical_region(c), cPartition);

}

void task_4(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion a = regions[0];
  PhysicalRegion b = regions[1];
  PhysicalRegion c = regions[2];

  task_4Args* args = (task_4Args*)(task->args);
  int32_t kios = args->kios;

  auto a_index_space = get_index_space(a);
  auto b_index_space = get_index_space(b);
  auto c_index_space = get_index_space(c);
  AccessorROdouble2 b_vals(b, FID_VAL);
  AccessorROdouble2 c_vals(c, FID_VAL);
  AccessorRWdouble2 a_vals(a, FID_VAL);

  auto aDomain = runtime->get_index_space_domain(ctx, a_index_space);
  auto bDomain = runtime->get_index_space_domain(ctx, b_index_space);
  auto cDomain = runtime->get_index_space_domain(ctx, c_index_space);
  if (bDomain.get_volume() == 0 || cDomain.get_volume() == 0)
    return ;

  double alpha = 1.0000000000000000;
  hipblasHandle_t handle = getCuBLAS();
  hipStream_t taskStream = hipStream_t();
  hipStreamCreate(&(taskStream));
  CHECK_CUBLAS(hipblasSetStream(handle, taskStream));
  CHECK_CUBLAS(hipblasDgemm(
    handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    (1 + (cDomain.hi()[1] - cDomain.lo()[1])),
    (1 + (bDomain.hi()[0] - bDomain.lo()[0])),
    (1 + (cDomain.hi()[0] - cDomain.lo()[0])),
    &(alpha),
    c_vals.ptr(cDomain.lo()),
    (c_vals.accessor.strides[0] / sizeof(double)),
    b_vals.ptr(bDomain.lo()),
    (b_vals.accessor.strides[0] / sizeof(double)),
    &(alpha),
    a_vals.ptr(aDomain.lo()),
    (a_vals.accessor.strides[0] / sizeof(double))
  ));
}

void task_5(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion a = regions[0];
  PhysicalRegion b = regions[1];
  PhysicalRegion c = regions[2];

  int32_t distFused1 = task->index_point[0];
  task_5Args* args = (task_5Args*)(task->args);
  int64_t aPartitionBounds0hi = args->aPartitionBounds0hi;
  int64_t aPartitionBounds0lo = args->aPartitionBounds0lo;
  int64_t aPartitionBounds1hi = args->aPartitionBounds1hi;
  int64_t aPartitionBounds1lo = args->aPartitionBounds1lo;
  int32_t b1_dimension = args->b1_dimension;
  int32_t b2_dimension = args->b2_dimension;
  int32_t c1_dimension = args->c1_dimension;
  int32_t c2_dimension = args->c2_dimension;
  int32_t gridX = args->gridX;
  int32_t in = args->in;
  int32_t jn = args->jn;
  int32_t kos = args->kos;

  auto b_index_space = get_index_space(b);
  auto c_index_space = get_index_space(c);

  int32_t iln = getIndexPoint(task, 0);
  int32_t jln = getIndexPoint(task, 1);
  Point<1> lowerBound = Point<1>(0);
  Point<1> upperBound = Point<1>(1);
  auto kiosIndexSpace = runtime->create_index_space(ctx, Rect<1>(lowerBound, upperBound));
  DomainT<1> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<1>(kiosIndexSpace));
  DomainPointColoring bColoring = DomainPointColoring();
  DomainPointColoring cColoring = DomainPointColoring();
  for (PointInDomainIterator<1> itr = PointInDomainIterator<1>(domain); itr.valid(); itr++) {
    int32_t kios = (*itr)[0];
    Point<2> bStart = Point<2>((iln * ((((aPartitionBounds0hi - aPartitionBounds0lo) + 1) + 1) / 2) + aPartitionBounds0lo), (((jn + (in + kos)) % gridX) * ((c1_dimension + (gridX - 1)) / gridX) + (((jln + (iln + kios)) % 2) * ((((c1_dimension + (gridX - 1)) / gridX - 0 / gridX) + 1) / 2) + (0 / gridX) / 2)));
    Point<2> bEnd = Point<2>(TACO_MIN(((iln * ((((aPartitionBounds0hi - aPartitionBounds0lo) + 1) + 1) / 2) + ((((aPartitionBounds0hi - aPartitionBounds0lo) + 1) + 1) / 2 - 1)) + aPartitionBounds0lo),(b1_dimension - 1)), TACO_MIN((((jn + (in + kos)) % gridX) * ((c1_dimension + (gridX - 1)) / gridX) + (((jln + (iln + kios)) % 2) * ((((c1_dimension + (gridX - 1)) / gridX - 0 / gridX) + 1) / 2) + (((c1_dimension + (gridX - 1)) / gridX + 1) / 2 - 1))),(b2_dimension - 1)));
    Rect<2> bRect = Rect<2>(bStart, bEnd);
    auto bDomain = runtime->get_index_space_domain(ctx, b_index_space);
    if (!bDomain.contains(bRect.lo) || !bDomain.contains(bRect.hi)) bRect = bRect.make_empty();

    bColoring[(*itr)] = bRect;
    Point<2> cStart = Point<2>((((jn + (in + kos)) % gridX) * ((c1_dimension + (gridX - 1)) / gridX) + (((jln + (iln + kios)) % 2) * ((((c1_dimension + (gridX - 1)) / gridX - 0 / gridX) + 1) / 2) + (0 / gridX) / 2)), (jln * ((((aPartitionBounds1hi - aPartitionBounds1lo) + 1) + 1) / 2) + aPartitionBounds1lo));
    Point<2> cEnd = Point<2>(TACO_MIN((((jn + (in + kos)) % gridX) * ((c1_dimension + (gridX - 1)) / gridX) + (((jln + (iln + kios)) % 2) * ((((c1_dimension + (gridX - 1)) / gridX - 0 / gridX) + 1) / 2) + (((c1_dimension + (gridX - 1)) / gridX + 1) / 2 - 1))),(c1_dimension - 1)), TACO_MIN(((jln * ((((aPartitionBounds1hi - aPartitionBounds1lo) + 1) + 1) / 2) + ((((aPartitionBounds1hi - aPartitionBounds1lo) + 1) + 1) / 2 - 1)) + aPartitionBounds1lo),(c2_dimension - 1)));
    Rect<2> cRect = Rect<2>(cStart, cEnd);
    auto cDomain = runtime->get_index_space_domain(ctx, c_index_space);
    if (!cDomain.contains(cRect.lo) || !cDomain.contains(cRect.hi)) cRect = cRect.make_empty();

    cColoring[(*itr)] = cRect;
  }
  auto bPartition = runtime->create_index_partition(ctx, b_index_space, domain, bColoring, LEGION_DISJOINT_KIND);
  auto cPartition = runtime->create_index_partition(ctx, c_index_space, domain, cColoring, LEGION_DISJOINT_KIND);
  for (PointInDomainIterator<1> itr = PointInDomainIterator<1>(domain); itr.valid(); itr++) {
    int32_t kios = (*itr);
    RegionRequirement aReq = RegionRequirement(get_logical_region(a), READ_WRITE, EXCLUSIVE, get_logical_region(a));
    aReq.add_field(FID_VAL);
    auto bsubReg = runtime->get_logical_subregion_by_color(ctx, runtime->get_logical_partition(ctx, get_logical_region(b), bPartition), kios);
    RegionRequirement bReq = RegionRequirement(bsubReg, READ_ONLY, EXCLUSIVE, get_logical_region(b));
    bReq.add_field(FID_VAL);
    auto csubReg = runtime->get_logical_subregion_by_color(ctx, runtime->get_logical_partition(ctx, get_logical_region(c), cPartition), kios);
    RegionRequirement cReq = RegionRequirement(csubReg, READ_ONLY, EXCLUSIVE, get_logical_region(c));
    cReq.add_field(FID_VAL);
    task_4Args taskArgsRaw;
    taskArgsRaw.kios = kios;
    TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_4Args));
    TaskLauncher launcher = TaskLauncher(taskID(4), taskArgs);
    launcher.add_region_requirement(aReq);
    launcher.add_region_requirement(bReq);
    launcher.add_region_requirement(cReq);
    runtime->execute_task(ctx, launcher);
  }

}

void task_6(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion a = regions[0];
  PhysicalRegion b = regions[1];
  PhysicalRegion c = regions[2];

  task_6Args* args = (task_6Args*)(task->args);
  int32_t a1_dimension = args->a1_dimension;
  int32_t a2_dimension = args->a2_dimension;
  int64_t aPartitionBounds0hi = args->aPartitionBounds0hi;
  int64_t aPartitionBounds0lo = args->aPartitionBounds0lo;
  int64_t aPartitionBounds1hi = args->aPartitionBounds1hi;
  int64_t aPartitionBounds1lo = args->aPartitionBounds1lo;
  int32_t b1_dimension = args->b1_dimension;
  int32_t b2_dimension = args->b2_dimension;
  int32_t c1_dimension = args->c1_dimension;
  int32_t c2_dimension = args->c2_dimension;
  int32_t gridX = args->gridX;
  int32_t in = args->in;
  int32_t jn = args->jn;
  int32_t kos = args->kos;

  auto a_index_space = get_index_space(a);

  Point<2> lowerBound = Point<2>(0, 0);
  Point<2> upperBound = Point<2>(1, 1);
  auto distFused1IndexSpace = runtime->create_index_space(ctx, Rect<2>(lowerBound, upperBound));
  DomainT<2> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<2>(distFused1IndexSpace));
  DomainPointColoring aColoring = DomainPointColoring();
  for (PointInDomainIterator<2> itr = PointInDomainIterator<2>(domain); itr.valid(); itr++) {
    int32_t iln = (*itr)[0];
    int32_t jln = (*itr)[1];
    Point<2> aStart = Point<2>((iln * ((((aPartitionBounds0hi - aPartitionBounds0lo) + 1) + 1) / 2) + aPartitionBounds0lo), (jln * ((((aPartitionBounds1hi - aPartitionBounds1lo) + 1) + 1) / 2) + aPartitionBounds1lo));
    Point<2> aEnd = Point<2>(TACO_MIN(((iln * ((((aPartitionBounds0hi - aPartitionBounds0lo) + 1) + 1) / 2) + ((((aPartitionBounds0hi - aPartitionBounds0lo) + 1) + 1) / 2 - 1)) + aPartitionBounds0lo),(a1_dimension - 1)), TACO_MIN(((jln * ((((aPartitionBounds1hi - aPartitionBounds1lo) + 1) + 1) / 2) + ((((aPartitionBounds1hi - aPartitionBounds1lo) + 1) + 1) / 2 - 1)) + aPartitionBounds1lo),(a2_dimension - 1)));
    Rect<2> aRect = Rect<2>(aStart, aEnd);
    auto aDomain = runtime->get_index_space_domain(ctx, a_index_space);
    if (!aDomain.contains(aRect.lo) || !aDomain.contains(aRect.hi)) aRect = aRect.make_empty();

    aColoring[(*itr)] = aRect;
  }
  auto aPartition = runtime->create_index_partition(ctx, a_index_space, domain, aColoring, LEGION_DISJOINT_KIND);
  LogicalPartition aLogicalPartition = runtime->get_logical_partition(ctx, get_logical_region(a), aPartition);
  RegionRequirement aReq = RegionRequirement(aLogicalPartition, 0, READ_WRITE, EXCLUSIVE, get_logical_region(a));
  aReq.add_field(FID_VAL);
  aReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
  RegionRequirement bReq = RegionRequirement(get_logical_region(b), READ_ONLY, EXCLUSIVE, get_logical_region(b));
  bReq.add_field(FID_VAL);
  bReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
  RegionRequirement cReq = RegionRequirement(get_logical_region(c), READ_ONLY, EXCLUSIVE, get_logical_region(c));
  cReq.add_field(FID_VAL);
  cReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
  task_5Args taskArgsRaw;
  taskArgsRaw.aPartitionBounds0hi = aPartitionBounds0hi;
  taskArgsRaw.aPartitionBounds0lo = aPartitionBounds0lo;
  taskArgsRaw.aPartitionBounds1hi = aPartitionBounds1hi;
  taskArgsRaw.aPartitionBounds1lo = aPartitionBounds1lo;
  taskArgsRaw.b1_dimension = b1_dimension;
  taskArgsRaw.b2_dimension = b2_dimension;
  taskArgsRaw.c1_dimension = c1_dimension;
  taskArgsRaw.c2_dimension = c2_dimension;
  taskArgsRaw.gridX = gridX;
  taskArgsRaw.in = in;
  taskArgsRaw.jn = jn;
  taskArgsRaw.kos = kos;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_5Args));
  IndexLauncher launcher = IndexLauncher(taskID(5), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(aReq);
  launcher.add_region_requirement(bReq);
  launcher.add_region_requirement(cReq);
  launcher.tag = launcher.tag | Mapping::DefaultMapper::SAME_ADDRESS_SPACE;
  runtime->execute_index_space(ctx, launcher);

}

void task_7(const Task* task, const std::vector<PhysicalRegion>& regions, Context ctx, Runtime* runtime) {
  PhysicalRegion a = regions[0];
  PhysicalRegion b = regions[1];
  PhysicalRegion c = regions[2];

  int32_t distFused = task->index_point[0];
  task_7Args* args = (task_7Args*)(task->args);
  int32_t a1_dimension = args->a1_dimension;
  int32_t a2_dimension = args->a2_dimension;
  int32_t b1_dimension = args->b1_dimension;
  int32_t b2_dimension = args->b2_dimension;
  int32_t c1_dimension = args->c1_dimension;
  int32_t c2_dimension = args->c2_dimension;
  int32_t gridX = args->gridX;

  auto a_index_space = get_index_space(a);
  auto b_index_space = get_index_space(b);
  auto c_index_space = get_index_space(c);

  int32_t in = getIndexPoint(task, 0);
  int32_t jn = getIndexPoint(task, 1);
  auto aPartitionBounds = runtime->get_index_space_domain(ctx, a_index_space);
  int64_t aPartitionBounds0lo = aPartitionBounds.lo()[0];
  int64_t aPartitionBounds0hi = aPartitionBounds.hi()[0];
  int64_t aPartitionBounds1lo = aPartitionBounds.lo()[1];
  int64_t aPartitionBounds1hi = aPartitionBounds.hi()[1];
  Point<1> lowerBound = Point<1>(0);
  Point<1> upperBound = Point<1>((gridX - 1));
  auto kosIndexSpace = runtime->create_index_space(ctx, Rect<1>(lowerBound, upperBound));
  DomainT<1> domain = runtime->get_index_space_domain(ctx, IndexSpaceT<1>(kosIndexSpace));
  DomainPointColoring bColoring = DomainPointColoring();
  DomainPointColoring cColoring = DomainPointColoring();
  for (PointInDomainIterator<1> itr = PointInDomainIterator<1>(domain); itr.valid(); itr++) {
    int32_t kos = (*itr)[0];
    Point<2> bStart = Point<2>(aPartitionBounds0lo, (((jn + (in + kos)) % gridX) * ((c1_dimension + (gridX - 1)) / gridX) + 0 / gridX));
    Point<2> bEnd = Point<2>(TACO_MIN(((((aPartitionBounds0hi - aPartitionBounds0lo) + 1) - 1) + aPartitionBounds0lo),(b1_dimension - 1)), TACO_MIN((((jn + (in + kos)) % gridX) * ((c1_dimension + (gridX - 1)) / gridX) + ((c1_dimension + (gridX - 1)) / gridX - 1)),(b2_dimension - 1)));
    Rect<2> bRect = Rect<2>(bStart, bEnd);
    auto bDomain = runtime->get_index_space_domain(ctx, b_index_space);
    if (!bDomain.contains(bRect.lo) || !bDomain.contains(bRect.hi)) bRect = bRect.make_empty();

    bColoring[(*itr)] = bRect;
    Point<2> cStart = Point<2>((((jn + (in + kos)) % gridX) * ((c1_dimension + (gridX - 1)) / gridX) + 0 / gridX), aPartitionBounds1lo);
    Point<2> cEnd = Point<2>(TACO_MIN((((jn + (in + kos)) % gridX) * ((c1_dimension + (gridX - 1)) / gridX) + ((c1_dimension + (gridX - 1)) / gridX - 1)),(c1_dimension - 1)), TACO_MIN(((((aPartitionBounds1hi - aPartitionBounds1lo) + 1) - 1) + aPartitionBounds1lo),(c2_dimension - 1)));
    Rect<2> cRect = Rect<2>(cStart, cEnd);
    auto cDomain = runtime->get_index_space_domain(ctx, c_index_space);
    if (!cDomain.contains(cRect.lo) || !cDomain.contains(cRect.hi)) cRect = cRect.make_empty();

    cColoring[(*itr)] = cRect;
  }
  auto bPartition = runtime->create_index_partition(ctx, b_index_space, domain, bColoring, LEGION_DISJOINT_KIND);
  auto cPartition = runtime->create_index_partition(ctx, c_index_space, domain, cColoring, LEGION_DISJOINT_KIND);
  for (PointInDomainIterator<1> itr = PointInDomainIterator<1>(domain); itr.valid(); itr++) {
    int32_t kos = (*itr);
    RegionRequirement aReq = RegionRequirement(get_logical_region(a), READ_WRITE, EXCLUSIVE, get_logical_region(a));
    aReq.add_field(FID_VAL);
    aReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
    auto bsubReg = runtime->get_logical_subregion_by_color(ctx, runtime->get_logical_partition(ctx, get_logical_region(b), bPartition), kos);
    RegionRequirement bReq = RegionRequirement(bsubReg, READ_ONLY, EXCLUSIVE, get_logical_region(b));
    bReq.add_field(FID_VAL);
    bReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
    auto csubReg = runtime->get_logical_subregion_by_color(ctx, runtime->get_logical_partition(ctx, get_logical_region(c), cPartition), kos);
    RegionRequirement cReq = RegionRequirement(csubReg, READ_ONLY, EXCLUSIVE, get_logical_region(c));
    cReq.add_field(FID_VAL);
    cReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
    task_6Args taskArgsRaw;
    taskArgsRaw.a1_dimension = a1_dimension;
    taskArgsRaw.a2_dimension = a2_dimension;
    taskArgsRaw.aPartitionBounds0hi = aPartitionBounds0hi;
    taskArgsRaw.aPartitionBounds0lo = aPartitionBounds0lo;
    taskArgsRaw.aPartitionBounds1hi = aPartitionBounds1hi;
    taskArgsRaw.aPartitionBounds1lo = aPartitionBounds1lo;
    taskArgsRaw.b1_dimension = b1_dimension;
    taskArgsRaw.b2_dimension = b2_dimension;
    taskArgsRaw.c1_dimension = c1_dimension;
    taskArgsRaw.c2_dimension = c2_dimension;
    taskArgsRaw.gridX = gridX;
    taskArgsRaw.in = in;
    taskArgsRaw.jn = jn;
    taskArgsRaw.kos = kos;
    TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_6Args));
    TaskLauncher launcher = TaskLauncher(taskID(6), taskArgs);
    launcher.add_region_requirement(aReq);
    launcher.add_region_requirement(bReq);
    launcher.add_region_requirement(cReq);
    runtime->execute_task(ctx, launcher);
  }

}

void computeLegion(Context ctx, Runtime* runtime, LogicalRegion a, LogicalRegion b, LogicalRegion c, LogicalPartition aPartition, int32_t gridX) {
  int a1_dimension = runtime->get_index_space_domain(get_index_space(a)).hi()[0] + 1;
  int a2_dimension = runtime->get_index_space_domain(get_index_space(a)).hi()[1] + 1;
  int b1_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[0] + 1;
  int b2_dimension = runtime->get_index_space_domain(get_index_space(b)).hi()[1] + 1;
  int c1_dimension = runtime->get_index_space_domain(get_index_space(c)).hi()[0] + 1;
  int c2_dimension = runtime->get_index_space_domain(get_index_space(c)).hi()[1] + 1;

  DomainT<2> domain = runtime->get_index_partition_color_space(ctx, get_index_partition(aPartition));
  for (PointInDomainIterator<2> itr = PointInDomainIterator<2>(domain); itr.valid(); itr++) {
    DomainPoint domPoint = (*itr);
    auto aPartitionBounds = runtime->get_index_space_domain(runtime->get_logical_subregion_by_color(ctx, aPartition, domPoint).get_index_space());
  }
  RegionRequirement aReq = RegionRequirement(aPartition, 0, READ_WRITE, EXCLUSIVE, get_logical_region(a));
  aReq.add_field(FID_VAL);
  aReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
  RegionRequirement bReq = RegionRequirement(get_logical_region(b), READ_ONLY, EXCLUSIVE, get_logical_region(b));
  bReq.add_field(FID_VAL);
  bReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
  RegionRequirement cReq = RegionRequirement(get_logical_region(c), READ_ONLY, EXCLUSIVE, get_logical_region(c));
  cReq.add_field(FID_VAL);
  cReq.tag = Mapping::DefaultMapper::VIRTUAL_MAP;
  task_7Args taskArgsRaw;
  taskArgsRaw.a1_dimension = a1_dimension;
  taskArgsRaw.a2_dimension = a2_dimension;
  taskArgsRaw.b1_dimension = b1_dimension;
  taskArgsRaw.b2_dimension = b2_dimension;
  taskArgsRaw.c1_dimension = c1_dimension;
  taskArgsRaw.c2_dimension = c2_dimension;
  taskArgsRaw.gridX = gridX;
  TaskArgument taskArgs = TaskArgument(&taskArgsRaw, sizeof(task_7Args));
  IndexLauncher launcher = IndexLauncher(taskID(7), domain, taskArgs, ArgumentMap());
  launcher.add_region_requirement(aReq);
  launcher.add_region_requirement(bReq);
  launcher.add_region_requirement(cReq);
  auto fm = runtime->execute_index_space(ctx, launcher);
  fm.wait_all_results();

}
void registerTacoTasks() {
  {
    TaskVariantRegistrar registrar(taskID(1), "task_1");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_1>(registrar, "task_1");
  }
  {
    TaskVariantRegistrar registrar(taskID(2), "task_2");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_2>(registrar, "task_2");
  }
  {
    TaskVariantRegistrar registrar(taskID(3), "task_3");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_3>(registrar, "task_3");
  }
  {
    TaskVariantRegistrar registrar(taskID(4), "task_4");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_leaf();
    Runtime::preregister_task_variant<task_4>(registrar, "task_4");
  }
  {
    TaskVariantRegistrar registrar(taskID(5), "task_5");
    registrar.add_constraint(ProcessorConstraint(Processor::TOC_PROC));
    registrar.set_inner();
    Runtime::preregister_task_variant<task_5>(registrar, "task_5");
  }
  {
    TaskVariantRegistrar registrar(taskID(6), "task_6");
    registrar.add_constraint(ProcessorConstraint(Processor::LOC_PROC));
    registrar.set_inner();
    Runtime::preregister_task_variant<task_6>(registrar, "task_6");
  }
  {
    TaskVariantRegistrar registrar(taskID(7), "task_7");
    registrar.add_constraint(ProcessorConstraint(Processor::LOC_PROC));
    registrar.set_inner();
    Runtime::preregister_task_variant<task_7>(registrar, "task_7");
  }
}
